#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(call)                                                                                                                                                     \
    {                                                                                                                                                                              \
        hipError_t err = call;                                                                                                                                                    \
        if (err != hipSuccess) {                                                                                                                                                  \
            fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", __FILE__, __LINE__, hipGetErrorString(err));                                                             \
            exit(EXIT_FAILURE);                                                                                                                                                    \
        }                                                                                                                                                                          \
    }

__global__ void concatenate(float *param_0, // Source tensor 1
                            float *param_1, // Source tensor 2
                            float *param_2,// Destination tensor
                            int c1,int c2, int c3        
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i=0; i< c1;i++) { // We have 4 warps processing 64 elements each
        
        // Calculate index for source and destination
        int src_idx = idx * c1+ i;
        int dest_idx = idx * c3+ i;

        float src_val = param_0[src_idx];
        param_2[dest_idx] = src_val;
    }
    for (int j=0; j< c2;j++) { // We have 4 warps processing 64 elements each
        // Calculate index for source and destination
        int src_idx = idx * c2 + j;
        int dest_idx = idx * c3 + c1+j;
        float src_val2 = param_1[src_idx];
        // Write values to destination tensor
        param_2[dest_idx] = src_val2;
        printf("%f ",param_2[dest_idx]);

    }
}


int main() {
    struct timeval start1, end1, start2, end2;
    gettimeofday(&start1, NULL);

    // 主机上创建矩阵
    float *A = (float *)malloc(sizeof(float) * 1024 * 1728);
    float *B = (float *)malloc(sizeof(float) * 1024* 192);
    float *C = (float *)malloc(sizeof(float) * 1024 * 1920);
    // malloc device memory cpu->gpu
    float *d_dataA, *d_dataB, *d_dataC;
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_dataA, sizeof(float) * 1024 * 1728));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_dataB, sizeof(float) * 1024* 192));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_dataC, sizeof(float) * 1024 * 1920));

    // set value随机初始化矩阵A和矩阵B
    for (int i = 0; i < 1024 * 1728; i++) {
        A[i] = i; // Random integers between 0 and 99
    }
    for (int i = 0; i < 1024* 192; i++) {
        B[i] = i; // Random integers between 0 and 99
    }
    for (int i = 0; i < 1024 * 1920; i++) {
        C[i] =0; // Random integers between 0 and 99
    }

    // 复制数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_dataA, A, sizeof(float) * 1024 * 1728, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_dataB, B, sizeof(float) * 1024* 192, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_dataC, C, sizeof(float) * 1024 * 1920, hipMemcpyHostToDevice));

    // 核函数配置
    dim3 threadPerBlock(64); // 每个块中的线程数
    dim3 blockNumber(16);     // 块数
    CHECK_CUDA_ERROR(hipGetLastError());
    //CHECK_CUDA_ERROR(cudaDeviceSynchronize());
    gettimeofday(&start2, NULL);
    CHECK_CUDA_ERROR(hipGetLastError());
    //CHECK_CUDA_ERROR(cudaDeviceSynchronize());
    // 调用核函数
    concatenate<<<blockNumber, threadPerBlock>>>(d_dataA, d_dataB, d_dataC, 1728,192, 1920);

    // 检查核函数调用的错误
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    gettimeofday(&end2, NULL);
    float timeuse_kernel = 1000000 * (end2.tv_sec - start2.tv_sec) + end2.tv_usec - start2.tv_usec;
    printf("Kernel execution time is %f ms\n", timeuse_kernel / 1000);

    // 拷贝计算结果到主机
    CHECK_CUDA_ERROR(hipMemcpy(C, d_dataC, sizeof(float) * 1024 * 1920, hipMemcpyDeviceToHost));

    // 输出结果
    printf("Result matrix first element: %f\n", C[100]);
    printf("Result matrix last element: %f\n", C[1024 * 1920 - 1]);
    // 释放内存
    free(A);
    free(B);
    free(C);
    CHECK_CUDA_ERROR(hipFree(d_dataA));
    CHECK_CUDA_ERROR(hipFree(d_dataB));
    CHECK_CUDA_ERROR(hipFree(d_dataC));

    gettimeofday(&end1, NULL);
    float timeuse = 1000000 * (end1.tv_sec - start1.tv_sec) + end1.tv_usec - start1.tv_usec;
    printf("Total time is %f ms\n", timeuse / 1000);

    return 0;
}
