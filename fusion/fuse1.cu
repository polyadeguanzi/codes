//fusion for concanate and gemm
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define CHECK_CUDA_ERROR(call)                                                                                                                                                     \
    {                                                                                                                                                                              \
        hipError_t err = call;                                                                                                                                                    \
        if (err != hipSuccess) {                                                                                                                                                  \
            fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", __FILE__, __LINE__, hipGetErrorString(err));                                                             \
            exit(EXIT_FAILURE);                                                                                                                                                    \
        }                                                                                                                                                                          \
    }

__global__ void concatenate(float *param_0, // Source tensor 1
                            float *param_1, // Source tensor 2
                            float *param_2,// Destination tensor
                            int c1,int c2, int c3        
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i=0; i< c1;i++) { // We have 4 warps processing 64 elements each
        
        // Calculate index for source and destination
        int src_idx = idx * c1+ i;
        int dest_idx = idx * c3+ i;

        float src_val = param_0[src_idx];
        param_2[dest_idx] = src_val;
    }
    for (int j=0; j< c2;j++) { // We have 4 warps processing 64 elements each
        // Calculate index for source and destination
        int src_idx = idx * c2 + j;
        int dest_idx = idx * c3 + c1+j;
        float src_val2 = param_1[src_idx];
        // Write values to destination tensor
        param_2[dest_idx] = src_val2;
        //printf("%f ",param_2[dest_idx]);

    }
}


__global__ void gemm(float *M, 
                     float *N, 
                     float *P, 
                     int width) 
{
    // 计算线性索引
    printf("%d",1);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d",idx);
    float sum = 0;
    for (int k = 0; k < width; k++) {
        printf("%f",M[1]);
        sum += M[idx*width + k] * N[k *64 + threadIdx.x];
        printf("%d ",sum);
        
    }
    for (int t=0;t<64;t++){
        P[idx*64+t]=sum;
    }
    
}

int main() {
    //struct timeval start1, end1, start2, end2;

    // 主机上创建矩阵
    float *A = (float *)malloc(sizeof(float) * 1024 * 1728);
    float *B = (float *)malloc(sizeof(float) * 1024* 192);
    float *C = (float *)malloc(sizeof(float) * 1024 * 1920);
    float *W = (float *)malloc(sizeof(float) * 1920 * 64);
    float *R = (float *)malloc(sizeof(float) * 1024 * 64);
    // malloc device memory cpu->gpu
    float *d_dataA, *d_dataB, *d_dataC, *d_W, *d_dataD;
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_dataA, sizeof(float) * 1024 * 1728));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_dataB, sizeof(float) * 1024* 192));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_dataC, sizeof(float) * 1024 * 1920));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_W, sizeof(float) * 1920 * 64));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_dataD, sizeof(float) * 1024 * 64));

    // set value随机初始化矩阵A和矩阵B
    for (int i = 0; i < 1024 * 1728; i++) {
        A[i] = i%10+1; // Random integers between 0 and 99
    }
    for (int i = 0; i < 1024* 192; i++) {
        B[i] = i%10+1; // Random integers between 0 and 99
    }
    for (int i = 0; i < 1024 * 1920; i++) {
        C[i] =0; // Random integers between 0 and 99
    }
    for (int i = 0; i < 1920 * 64; i++) {
        W[i] =0; // Random integers between 0 and 99
    }
    for (int i = 0; i < 1024 * 64; i++) {
        R[i] =0; // Random integers between 0 and 99
    }

    // 复制数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_dataA, A, sizeof(float) * 1024 * 1728, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_dataB, B, sizeof(float) * 1024* 192, hipMemcpyHostToDevice));
    //CHECK_CUDA_ERROR(cudaMemcpy(d_dataC, C, sizeof(float) * 1024 * 1920, cudaMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_W, W, sizeof(float) * 1920 * 64, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_dataD, R, sizeof(float) * 1024 * 64, hipMemcpyHostToDevice));

    // 核函数配置
    dim3 threadPerBlock1(64); // 每个块中的线程数
    dim3 blockNumber1(16);     // 块数
    dim3 threadPerBlock2(64); // 每个块中的线程数
    dim3 blockNumber2(16);     // 块数

    concatenate<<<blockNumber1, threadPerBlock1>>>(d_dataA, d_dataB, d_dataC, 1728,192, 1920);

    // 检查核函数调用的错误
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    // 拷贝计算结果到主机
    CHECK_CUDA_ERROR(hipMemcpy(C, d_dataC, sizeof(float) * 1024 * 1920, hipMemcpyDeviceToHost));
    printf("Result matrix first element: %f\n", C[100]);
    printf("Result matrix last element: %f\n", C[1024 * 1920 - 1]);
    CHECK_CUDA_ERROR(hipMemcpy(d_dataC, C, sizeof(float) * 1024 * 1920, hipMemcpyHostToDevice));//TODO：中间过程优化
    gemm<<<blockNumber2, threadPerBlock2>>>(d_dataC, d_W, d_dataD, 1920);

    
    CHECK_CUDA_ERROR(hipMemcpy(d_dataD, R, sizeof(float) * 1024 * 64, hipMemcpyDeviceToHost))
    // 输出结果
    printf("Result matrix first element: %f\n", R[100]);
    printf("Result matrix last element: %f\n", R[1024 * 64 - 1]);
    // 释放内存
    free(A);
    free(B);
    free(C);
    free(W);
    free(R);
    CHECK_CUDA_ERROR(hipFree(d_dataA));
    CHECK_CUDA_ERROR(hipFree(d_dataB));
    CHECK_CUDA_ERROR(hipFree(d_dataC));
    CHECK_CUDA_ERROR(hipFree(d_W));
    CHECK_CUDA_ERROR(hipFree(d_dataD));




    return 0;
}
